#include "hip/hip_runtime.h"
#pragma once
#include "mb.cuh"
#include <opencv2/opencv.hpp>
#include <opencv2/cudev.hpp>
#include <hip/hip_runtime.h>

//for __syncthreads() 
#ifndef __HIPCC__ 
#define __HIPCC__ 
#endif 
#ifndef __CUDACC_RTC__ 
#define __CUDACC_RTC__ 
#endif //!(__CUDACC_RTC__) 
#include <>

#include "mb_param.h"



__global__ void myKernel(cv::cudev::PtrStepSz<uchar3> dst) {
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	if ((x < dst.cols) && (y < dst.rows)) {
		
		//���_���W
		double oy = (double)(dst.rows) / 2.0;
		double ox = (double)(dst.cols) / 3.0;

		//��ƂȂ镡�f��
		double nx = ((double)x - 2 * ox) / ox;
		double ny = ((double)y - oy) / oy;

		//����v�Z���ʂ̕��f��
		double zx = 0;
		double zy = 0;

		//����������
		double convergenceDecision = true;

		//�v�Z�p�e���v���[�g
		double tx = 0;
		double ty = 0;

		//���U���x�i�傫���قǑ������U�j
		int n = 0;
		
		for (int i = 0; i <= INF; i++) {

			double a = sqrt(zx * zx + zy * zy);
			if (a >= LIMIT) {
				convergenceDecision = false;
				break;
			}

			tx = zx * zx - zy * zy + nx;
			ty = 2 * zx * zy + ny;

			zx = tx;
			zy = ty;

			n = i;
		}

		//���U���x�ɉ����ĐF��ς���
		int r = (n % (INF / 11)) * 20;
		int g = (n % (INF / 15)) * 15;
		int b = (n % (INF / 19)) * 12;

		if (convergenceDecision) {
			//���������牽�����Ȃ��i�F�����j
		} else {
			dst.ptr(y)[x] = make_uchar3(b, g, r);
		}
	}
	//__syncthreads();
}


void createMB(cv::cuda::GpuMat &mat) {
	const dim3 block(32, 8);
	const dim3 grid(cv::cudev::divUp(mat.cols, block.x), cv::cudev::divUp(mat.rows, block.y));



	// ����CUDA�J�[�l�����Ăяo��
	myKernel << <grid, block >> > (mat);

	CV_CUDEV_SAFE_CALL(hipGetLastError());
	CV_CUDEV_SAFE_CALL(hipDeviceSynchronize());
}